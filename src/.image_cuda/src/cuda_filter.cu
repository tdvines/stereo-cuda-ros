#include "hip/hip_runtime.h"
#include "../include/cuda_filter.h"

__global__ void depthMapToPointCloud(const unsigned char* inputImage1, const unsigned char* inputImage2,
                                     pcl::PointCloud<pcl::PointXYZ>::Ptr point_cloud, int width, int height,
                                     float focal_length, float baseline) {
    // int x = blockIdx.x * blockDim.x + threadIdx.x;
    // int y = blockIdx.y * blockDim.y + threadIdx.y;

    // if (x < width && y < height) {
        // int index = y * width + x;
        // int index3 = index * 3; // index for RGB channels

        // unsigned char blue1 = inputImage1[index3];
        // unsigned char green1 = inputImage1[index3 + 1];
        // unsigned char red1 = inputImage1[index3 + 2];

        // unsigned char blue2 = inputImage2[index3];
        // unsigned char green2 = inputImage2[index3 + 1];
        // unsigned char red2 = inputImage2[index3 + 2];

        // // Convert to grayscale using luminance formula (Y = 0.299*R + 0.587*G + 0.114*B)
        // float gray1 = 0.299f * red1 + 0.587f * green1 + 0.114f * blue1;
        // float gray2 = 0.299f * red2 + 0.587f * green2 + 0.114f * blue2;

        // // Average the grayscale pixel values
        // float disparity = fabs(gray1 - gray2);
        // float depth = (baseline * focal_length) / disparity;

        // // Calculate the 3D coordinates of the point
        // float X = ((float)x - (float)width / 2.0f) * depth / focal_length;
        // float Y = ((float)y - (float)height / 2.0f) * depth / focal_length;
        // float Z = depth;

        // // Write point to point cloud
        // pcl::PointXYZ point;
        // point.x = X;
        // point.y = Y;
        // point.z = Z;
        // point_cloud->push_back(point);
    // }
}

pcl::PointCloud<pcl::PointXYZ>::Ptr depthMapToPointCloudMsg(const cv::Mat& inputImage1, const cv::Mat& inputImage2,
                                                             float focal_length, float baseline) {
    int width = inputImage1.cols;
    int height = inputImage1.rows;

    // // Allocate GPU memory for the input images
    // unsigned char* d_inputImage1 = nullptr;
    // unsigned char* d_inputImage2 = nullptr;
    // hipMalloc((void**)&d_inputImage1, width * height * 3 * sizeof(unsigned char));
    // hipMalloc((void**)&d_inputImage2, width * height * 3 * sizeof(unsigned char));
    // hipMemcpy(d_inputImage1, inputImage1.data, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
    // hipMemcpy(d_inputImage2, inputImage2.data, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

    // // Allocate GPU memory for the point cloud data
    // float* d_point_cloud_data = nullptr;
    // hipMalloc((void**)&d_point_cloud_data, width * height * 3 * sizeof(float));

    // // Set up grid and block dimensions
    // dim3 blockDim(16, 16);
    // dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // // Launch the CUDA kernel
    // depthMapToPointCloud<<<gridDim, blockDim>>>(d_inputImage1, d_inputImage2, d_point_cloud_data, width, height,
    //                                             focal_length, baseline);

    // // Copy the result back from GPU to CPU
    // float* h_point_cloud_data = new float[width * height * 3];
    // hipMemcpy(h_point_cloud_data, d_point_cloud_data, width * height * 3 * sizeof(float), hipMemcpyDeviceToHost);

    // // Convert to PCL point cloud
    pcl::PointCloud<pcl::PointXYZ>::Ptr point_cloud(new pcl::PointCloud<pcl::PointXYZ>);
    // for (int i = 0; i < width * height; ++i) {
    //     pcl::PointXYZ point;
    //     point.x = h_point_cloud_data[i * 3];
    //     point.y = h_point_cloud_data[i * 3 + 1];
    //     point.z = h_point_cloud_data[i * 3 + 2];
    //     point_cloud->push_back(point);
    // }

    // // Free GPU memory
    // hipFree(d_inputImage1);
    // hipFree(d_inputImage2);
    // hipFree(d_point_cloud_data);
    // delete[] h_point_cloud_data;

    return point_cloud;
}

int main()
{
    // // Load an image using OpenCV
    // cv::Mat inputImage = cv::imread("input.jpg");

    // if (inputImage.empty())
    // {
    //     std::cerr << "Error: Could not load input image." << std::endl;
    //     return -1;
    // }

    // Call the image processing function
    // cv::Mat filteredImage = processImage(inputImage);

    // // Display or save the filtered image
    // cv::imwrite("output.jpg", filteredImage);
    // cv::imshow("Filtered Image", filteredImage);
    // cv::waitKey(0);

    return 0;
}